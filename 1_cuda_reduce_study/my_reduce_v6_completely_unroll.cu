
#include <hip/hip_runtime.h>
#include <cstdio>

#define THREAD_PER_BLOCK 256

__device__ void warpReduce(volatile float *cache, unsigned int tid)
{
    cache[tid] += cache[tid + 32];
    //__syncthreads();
    cache[tid] += cache[tid + 16];
    //__syncthreads();
    cache[tid] += cache[tid + 8];
    //__syncthreads();
    cache[tid] += cache[tid + 4];
    //__syncthreads();
    cache[tid] += cache[tid + 2];
    //__syncthreads();
    cache[tid] += cache[tid + 1];
    //__syncthreads();
}
__global__ void reduce(float *d_input, float *d_output)
{
    int tid = threadIdx.x;
    volatile __shared__ float shared[THREAD_PER_BLOCK];

    float *input_begin = d_input + blockDim.x * blockIdx.x * 2;
    shared[tid] = input_begin[tid] + input_begin[tid + blockDim.x];
    __syncthreads();
    // #pragma unroll // 展开
    //     for (int i = blockDim.x / 2; i > 32; i /= 2)
    //     {
    //         if (tid < i)
    //         {
    //             shared[tid] += shared[tid + i];
    //         }
    //         __syncthreads();
    //     }

    if (tid < 128)
    {
        shared[tid] += shared[tid + 128];
    }
    __syncthreads();
    if (tid < 64)
    {
        shared[tid] += shared[tid + 64];
    }
    __syncthreads();

    if (tid < 32)
    {
        warpReduce(shared, tid);
    }

    if (tid == 0)
        d_output[blockIdx.x] = shared[0];
}

bool check(float *out, float *res, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (abs(out[i] - res[i]) > 1e-3)
            return false;
    }
    return true;
}

int main()
{
    const int N = 32 * 1024 * 1024;
    float *input = (float *)malloc(N * sizeof(float));
    float *d_input;
    hipMalloc((void **)&d_input, N * sizeof(float));

    int block_num = N / THREAD_PER_BLOCK / 2;
    float *output = (float *)malloc(block_num * sizeof(float));
    float *d_output;
    hipMalloc((void **)&d_output, block_num * sizeof(float));
    float *result = (float *)malloc(block_num * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        input[i] = 2.0 * (float)drand48() - 1.0;
    }

    for (int i = 0; i < block_num; i++)
    {
        float cur = 0;
        for (int j = 0; j < 2 * THREAD_PER_BLOCK; j++)
        {
            cur += input[2 * i * THREAD_PER_BLOCK + j];
        }
        result[i] = cur;
    }

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(block_num, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);

    reduce<<<Grid, Block>>>(d_input, d_output);

    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);
    if (check(output, result, block_num))
        printf("the ans is right\n");
    else
    {
        printf("the ans is wrong\n");
        for (int i = 0; i < block_num; i++)
        {
            printf("%lf ", output[i]);
        }
        printf("\n");
    }
    hipFree(d_input);
    hipFree(d_output);
}