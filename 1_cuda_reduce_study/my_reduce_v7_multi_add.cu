
#include <hip/hip_runtime.h>
#include <cstdio>

#define THREAD_PER_BLOCK 256

__device__ void warpReduce(volatile float *cache, unsigned int tid)
{
    cache[tid] += cache[tid + 32];
    //__syncthreads();
    cache[tid] += cache[tid + 16];
    //__syncthreads();
    cache[tid] += cache[tid + 8];
    //__syncthreads();
    cache[tid] += cache[tid + 4];
    //__syncthreads();
    cache[tid] += cache[tid + 2];
    //__syncthreads();
    cache[tid] += cache[tid + 1];
    //__syncthreads();
}
template <unsigned int NUM_PER_BLOCK, unsigned int NUM_PER_THREAD>
__global__ void reduce(float *d_input, float *d_output)
{
    int tid = threadIdx.x;
    __shared__ float shared[THREAD_PER_BLOCK];

    float *input_begin = d_input + NUM_PER_BLOCK * blockIdx.x;
    shared[tid] = 0;
    for (int i = 0; i < NUM_PER_THREAD; i++)
    {
        shared[tid] += input_begin[tid + i * THREAD_PER_BLOCK];
    }
    __syncthreads();

    // #pragma unroll // 展开
    //     for (int i = blockDim.x / 2; i > 32; i /= 2)
    //     {
    //         if (tid < i)
    //         {
    //             shared[tid] += shared[tid + i];
    //         }
    //         __syncthreads();
    //     }

    // THREAD_PER_BLOCK应该时编译器可以识别
    if (THREAD_PER_BLOCK >= 512)
    {
        if (tid < 256)
        {
            shared[tid] += shared[tid + 256];
        }
        __syncthreads();
    }
    if (THREAD_PER_BLOCK >= 256)
    {
        if (tid < 128)
        {
            shared[tid] += shared[tid + 128];
        }
        __syncthreads();
    }
    if (THREAD_PER_BLOCK >= 128)
    {
        if (tid < 64)
        {
            shared[tid] += shared[tid + 64];
        }
        __syncthreads();
    }

    if (tid < 32)
    {
        warpReduce(shared, tid);
    }

    if (tid == 0)
        d_output[blockIdx.x] = shared[0];
}

bool check(float *out, float *res, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (abs(out[i] - res[i]) > 0.005)
            return false;
    }
    return true;
}

int main()
{
    const int N = 32 * 1024 * 1024;
    float *input = (float *)malloc(N * sizeof(float));
    float *d_input;
    hipMalloc((void **)&d_input, N * sizeof(float));

    constexpr int block_num = 1024;
    constexpr int num_per_block = N / block_num;
    constexpr int num_per_thread = num_per_block / THREAD_PER_BLOCK;
    float *output = (float *)malloc(block_num * sizeof(float));
    float *d_output;
    hipMalloc((void **)&d_output, block_num * sizeof(float));
    float *result = (float *)malloc(block_num * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        input[i] = 2.0 * (float)drand48() - 1.0;
    }

    for (int i = 0; i < block_num; i++)
    {
        float cur = 0;
        for (int j = 0; j < num_per_block; j++)
        {
            cur += input[i * num_per_block + j];
        }
        result[i] = cur;
    }

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(block_num, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);

    reduce<num_per_block, num_per_thread><<<Grid, Block>>>(d_input, d_output);

    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);
    if (check(output, result, block_num))
        printf("the ans is right\n");
    else
    {
        printf("the ans is wrong\n");
        for (int i = 0; i < block_num; i++)
        {
            printf("%lf ", output[i]);
        }
        printf("\n");
    }
    hipFree(d_input);
    hipFree(d_output);
}