
#include <hip/hip_runtime.h>
#include <cstdio>

#define THREAD_PER_BLOCK 256

__global__ void reduce(float *d_input, float *d_output)
{

    __shared__ float shared[THREAD_PER_BLOCK];

    float *input_begin = d_input + blockDim.x * blockIdx.x * 2;
    shared[threadIdx.x] = input_begin[threadIdx.x] + input_begin[threadIdx.x + blockDim.x];
    __syncthreads();
    for (int i = blockDim.x / 2; i > 0; i /= 2)
    {
        if (threadIdx.x < i)
        {
            shared[threadIdx.x] += shared[threadIdx.x + i];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        d_output[blockIdx.x] = shared[0];
}

bool check(float *out, float *res, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (abs(out[i] - res[i]) > 1e-3)
            return false;
    }
    return true;
}

int main()
{
    const int N = 32 * 1024 * 1024;
    float *input = (float *)malloc(N * sizeof(float));
    float *d_input;
    hipMalloc((void **)&d_input, N * sizeof(float));

    int block_num = N / THREAD_PER_BLOCK / 2;
    float *output = (float *)malloc(block_num * sizeof(float));
    float *d_output;
    hipMalloc((void **)&d_output, block_num * sizeof(float));
    float *result = (float *)malloc(block_num * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        input[i] = 2.0 * (float)drand48() - 1.0;
    }

    for (int i = 0; i < block_num; i++)
    {
        float cur = 0;
        for (int j = 0; j < 2 * THREAD_PER_BLOCK; j++)
        {
            cur += input[2 * i * THREAD_PER_BLOCK + j];
        }
        result[i] = cur;
    }

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(block_num, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);

    reduce<<<Grid, Block>>>(d_input, d_output);

    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);
    if (check(output, result, block_num))
        printf("the ans is right\n");
    else
    {
        printf("the ans is wrong\n");
        for (int i = 0; i < block_num; i++)
        {
            printf("%lf ", output[i]);
        }
        printf("\n");
    }
    hipFree(d_input);
    hipFree(d_output);
}